#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/*inline void CUDA_ERROR_CHECK(const cudaError_t &err){
	if(err != cudaSuccess){
		fprintf(stderr, "CUDA error: %s\n", cudaGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}*/

__device__ int mandel(float c_re, float c_im, int maxIteration)
{
	float z_re = c_re, z_im = c_im;
    float new_re, new_im;

	int i = 0;
	while(i < maxIteration)
	{
		if (z_re * z_re + z_im * z_im > 4.f)
		    break;

		new_re = z_re * z_re - z_im * z_im;
		new_im = 2.f * z_re * z_im;
		z_re = c_re + new_re;
		z_im = c_im + new_im;

		++i;

		if (z_re * z_re + z_im * z_im > 4.f)
		    break;

		new_re = z_re * z_re - z_im * z_im;
		new_im = 2.f * z_re * z_im;
		z_re = c_re + new_re;
		z_im = c_im + new_im;

		++i;

		if (z_re * z_re + z_im * z_im > 4.f)
		    break;

		new_re = z_re * z_re - z_im * z_im;
		new_im = 2.f * z_re * z_im;
		z_re = c_re + new_re;
		z_im = c_im + new_im;

		++i;

		if (z_re * z_re + z_im * z_im > 4.f)
		    break;

		new_re = z_re * z_re - z_im * z_im;
		new_im = 2.f * z_re * z_im;
		z_re = c_re + new_re;
		z_im = c_im + new_im;

		++i;

		if (z_re * z_re + z_im * z_im > 4.f)
		    break;

		new_re = z_re * z_re - z_im * z_im;
		new_im = 2.f * z_re * z_im;
		z_re = c_re + new_re;
		z_im = c_im + new_im;

		++i;

		if (z_re * z_re + z_im * z_im > 4.f)
		    break;

		new_re = z_re * z_re - z_im * z_im;
		new_im = 2.f * z_re * z_im;
		z_re = c_re + new_re;
		z_im = c_im + new_im;

		++i;

		if (z_re * z_re + z_im * z_im > 4.f)
		    break;

		new_re = z_re * z_re - z_im * z_im;
		new_im = 2.f * z_re * z_im;
		z_re = c_re + new_re;
		z_im = c_im + new_im;

		++i;

		if (z_re * z_re + z_im * z_im > 4.f)
		    break;

		new_re = z_re * z_re - z_im * z_im;
		new_im = 2.f * z_re * z_im;
		z_re = c_re + new_re;
		z_im = c_im + new_im;

		++i;
	}

	return i;
}


__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int *d_res, int resX, int resY, int maxIterations){
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    int now_x = blockIdx.x * blockDim.x + threadIdx.x;
	int now_y = blockIdx.y * blockDim.y + threadIdx.y;

	if(now_x < resX || now_y < resY) {
		float x = lowerX + now_x * stepX;
		float y = lowerY + now_y * stepY;
		int idx = now_y * resX + now_x;
		d_res[idx] = mandel(x, y, maxIterations);
	}
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
	// float stepX = (upperX - lowerX) / resX;
	// float stepY = (upperY - lowerY) / resY;

	// int blocksX = ;
	// int blocksY = ;

	dim3 block(16, 16);
	dim3 grid((int) ceil(resX/16.0), (int) ceil(resY/16.0));

	int *d_res;
	int size = resX * resY * sizeof(int);

	hipMalloc((void**)&d_res, size);
	//int *h = (int*)malloc(size);
	
	mandelKernel <<< grid, block >>> (lowerX, lowerY, (upperX - lowerX) / resX, (upperY - lowerY) / resY, d_res, resX, resY, maxIterations);
	
	hipMemcpy(img, d_res, size, hipMemcpyDeviceToHost);
	//memcpy(img, h, size);

	//free(h);
	hipFree(d_res);
}